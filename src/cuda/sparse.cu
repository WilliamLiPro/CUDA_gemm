#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// template <int BLOCK_SIZE> __global__ void MatrixMulCUDA5( 
//     float * __restrict__ A_Val,
//     int* __restrict__ A_col_idx,
//     int* __restrict__ A_row_ptr,
//     float * __restrict__ B,
//     float * __restrict__ C, 
//     const int K,
//     const int N) {
//     // Block index
//     int bx = blockIdx.x;
//     int by = blockIdx.y;

//     // Thread index
//     int tx = threadIdx.x;
//     int ty = threadIdx.y;
    
//     float4 Csub[4] = {
//         {0, 0, 0, 0},
//         {0, 0, 0, 0},
//         {0, 0, 0, 0},
//         {0, 0, 0, 0}};
    
//     int row_ptr_start = A_row_ptr[by];
//     int row_ptr_end = A_row_ptr[by + 1];
    
//     for (int row_ptr = row_ptr_start ; row_ptr < row_ptr_end ; row_ptr = row_ptr + 1) {
//         int tile_idx = A_col_idx[row_ptr];
//         __shared__ float As[BLOCK_SIZE * BLOCK_SIZE];
//         __shared__ float Bs[BLOCK_SIZE * BLOCK_SIZE];
//         float* A = A_Val + BLOCK_SIZE * BLOCK_SIZE * row_ptr;
//         #pragma unroll
//         for ( int i = 0 ; i < 4 ; i ++ ) {
//             reinterpret_cast<float4*>(As + BLOCK_SIZE * (ty * 4 + i) + tx * 4)[0] 
//                 = reinterpret_cast<float4*>( A + BLOCK_SIZE * (ty * 4 + i) + tx * 4 )[0];
            
//                 reinterpret_cast<float4*>(Bs + BLOCK_SIZE * (ty * 4 + i) + tx * 4)[0] 
//                 = reinterpret_cast<float4*>(B + (BLOCK_SIZE * tile_idx + ty * 4 + i ) * N + BLOCK_SIZE * bx + tx * 4 )[0];
//         }
    
//         __syncthreads();

//         #pragma unroll
//         for (int k = 0; k < BLOCK_SIZE; ++k) {
            
//             Csub[0].x = fma(As[ty * 4 * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4], Csub[0].x);
//             Csub[0].y = fma(As[ty * 4 * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 1], Csub[0].y);
//             Csub[0].z = fma(As[ty * 4 * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 2], Csub[0].z);
//             Csub[0].w = fma(As[ty * 4 * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 3], Csub[0].w);
//             Csub[1].x = fma(As[(ty * 4 + 1) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4], Csub[1].x);
//             Csub[1].y = fma(As[(ty * 4 + 1) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 1], Csub[1].y);
//             Csub[1].z = fma(As[(ty * 4 + 1) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 2], Csub[1].z);
//             Csub[1].w = fma(As[(ty * 4 + 1) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 3], Csub[1].w);
//             Csub[2].x = fma(As[(ty * 4 + 2) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4], Csub[2].x);
//             Csub[2].y = fma(As[(ty * 4 + 2) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 1], Csub[2].y);
//             Csub[2].z = fma(As[(ty * 4 + 2) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 2], Csub[2].z);
//             Csub[2].w = fma(As[(ty * 4 + 2) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 3], Csub[2].w);
//             Csub[3].x = fma(As[(ty * 4 + 3) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4], Csub[3].x);
//             Csub[3].y = fma(As[(ty * 4 + 3) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 1], Csub[3].y);
//             Csub[3].z = fma(As[(ty * 4 + 3) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 2], Csub[3].z);
//             Csub[3].w = fma(As[(ty * 4 + 3) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 3], Csub[3].w);
            
//         }
//         // wait threads to finish , otherwise next tile will overwrite the shared memory
//         __syncthreads();
//     }

//     reinterpret_cast<float4*> (C + N * ( BLOCK_SIZE * by + ty * 4 ) + BLOCK_SIZE * bx + tx * 4 )[0] = Csub[0];
//     reinterpret_cast<float4*> (C + N * ( BLOCK_SIZE * by + ty * 4 + 1) + BLOCK_SIZE * bx + tx * 4 )[0] = Csub[1];
//     reinterpret_cast<float4*> (C + N * ( BLOCK_SIZE * by + ty * 4 + 2) + BLOCK_SIZE * bx + tx * 4 )[0] = Csub[2];
//     reinterpret_cast<float4*> (C + N * ( BLOCK_SIZE * by + ty * 4 + 3) + BLOCK_SIZE * bx + tx * 4 )[0] = Csub[3];
// }

// cal offset from row col and ld , in row-major matrix, ld is the width of the matrix
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// transfer float4
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])
template <
    const int BLOCK_SIZE_M,  // width of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // height of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // height of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X,  // width of block of C that each thread calculate
    const bool ENABLE_DOUBLE_BUFFER // whether enable double buffering or not
    > 
__global__ void MatrixMulCUDA5( 
    float * __restrict__ A_Val,
    int* __restrict__ A_col_idx,
    int* __restrict__ A_row_ptr,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int K,
    const int N) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // size of thread block
    const int bszx = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int bszy = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = bszy * bszx;

    // thread id
    const int tid = ty * bszx + tx;

    __shared__ float As[BLOCK_SIZE_M][BLOCK_SIZE_K]; // avoid bank conflict
    __shared__ float Bs[BLOCK_SIZE_K][BLOCK_SIZE_N];

    // registers for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X] = {0};
    // registers for A and B
    float frag_a[THREAD_SIZE_Y];
    float frag_b[THREAD_SIZE_X];
    
    // threads needed to load one row of tile
    // / 4 is because float4 is used
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;
    
    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4;
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;
    
    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;
    
    int row_ptr_start = A_row_ptr[by];
    int row_ptr_end = A_row_ptr[by + 1];
    
    for (int row_ptr = row_ptr_start ; row_ptr < row_ptr_end ; row_ptr = row_ptr + 1) {
        int tile_idx = A_col_idx[row_ptr] * BLOCK_SIZE_K;
        float* A = A_Val + BLOCK_SIZE_M * BLOCK_SIZE_K * row_ptr;
        // load A from global memory to shared memory
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
            FETCH_FLOAT4(As[A_TILE_ROW_START + i][A_TILE_COL]) = FETCH_FLOAT4(A[OFFSET(
                    A_TILE_ROW_START + i, // row
                    A_TILE_COL, // col
                    BLOCK_SIZE_K )]);
        }

        // load B from global memory to shared memory
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
            FETCH_FLOAT4(Bs[B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                    tile_idx + B_TILE_ROW_START + i, // row
                    B_TILE_COL + BLOCK_SIZE_N * bx, // col
                    K )]);
        }
    
        __syncthreads();

        // compute c
        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE_K; ++ k) {
            // load A from shared memory to register
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                frag_a[thread_y] = As[ty * THREAD_SIZE_Y + thread_y][k];
            }

            // load B from shared memory to register
            #pragma unroll
            for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
                FETCH_FLOAT4(frag_b[thread_x]) = FETCH_FLOAT4(Bs[k][THREAD_SIZE_X * tx + thread_x]);
            }
            
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                #pragma unroll
                for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                    accum[thread_y][thread_x] += frag_a[thread_y] * frag_b[thread_x];
                }
            }
            
        }
        __syncthreads();
    }

    // store back to C
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
            C[OFFSET(
                BLOCK_SIZE_M * by + ty * THREAD_SIZE_Y + thread_y,
                BLOCK_SIZE_N * bx + tx * THREAD_SIZE_X + thread_x,
                N)] = accum[thread_y][thread_x];
        }
    }
}